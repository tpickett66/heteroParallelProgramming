#include "hip/hip_runtime.h"
// MP 1
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//#include	<wb.h>

__global__ void vecAdd(float * in1, float * in2, float * out, int len) {
  int idx = threadIdx.x + blockDim.x * blockIdx.x;
  if(idx < len) {
    out[idx] = in1[idx] + in2[idx];
  }
}

int main(int argc, char ** argv) {
    wbArg_t args;
    int inputLength;
    float * hostInput1;
    float * hostInput2;
    float * hostOutput;
    float * deviceInput1;
    float * deviceInput2;
    float * deviceOutput;

    args = wbArg_read(argc, argv);

    wbTime_start(Generic, "Importing data and creating memory on host");
    hostInput1 = (float *) wbImport(wbArg_getInputFile(args, 0), &inputLength);
    hostInput2 = (float *) wbImport(wbArg_getInputFile(args, 1), &inputLength);
    hostOutput = (float *) malloc(inputLength * sizeof(float));
    wbTime_stop(Generic, "Importing data and creating memory on host");

    wbLog(TRACE, "The input length is ", inputLength);

    wbTime_start(GPU, "Allocating GPU memory.");

    hipMalloc((void **) &deviceInput1, &inputLength);
    hipMalloc((void **) &deviceInput2, &inputLength);
    hipMalloc((void **) &deviceOutput, &inputLength);

    wbTime_stop(GPU, "Allocating GPU memory.");

    wbTime_start(GPU, "Copying input memory to the GPU.");
    hipMemcpy(deviceInput1, hostInput1, inputLength, hipMemcpyHostToDevice);
    hipMemcpy(deviceInput2, hostInput2, inputLength, hipMemcpyHostToDevice);

    wbTime_stop(GPU, "Copying input memory to the GPU.");

    int blockSize = 256;
    struct dim3 DimGrid((inputLength - 1)/blockSize + 1, 1, 1);
    struct dim3 DimBlock(blockSize, 1, 1);

    wbTime_start(Compute, "Performing CUDA computation");
    vecAdd<<DimGrid, DimBlock>>(deviceInput1, deviceInput2, deviceOutput, inputLength);

    hipDeviceSynchronize();
    wbTime_stop(Compute, "Performing CUDA computation");

    wbTime_start(Copy, "Copying output memory to the CPU");
    hipMemcpy(hostOutput, deviceOutput, inputLength, hipMemcpyDeviceToHost);

    wbTime_stop(Copy, "Copying output memory to the CPU");

    wbTime_start(GPU, "Freeing GPU Memory");
    hipFree(deviceInput1);
    hipFree(deviceInput2);
    hipFree(deviceOutput);

    wbTime_stop(GPU, "Freeing GPU Memory");

    wbSolution(args, hostOutput, inputLength);

    free(hostInput1);
    free(hostInput2);
    free(hostOutput);

    return 0;
}


